#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "support.h"


const int threads = 256;

void memset_rand(uint8 *buf, int N) {
  for (int i=0; i<N; i++)
    buf[i] = rand() & 0xFF;
}

void histogram(int *hist, uint8 *buff, int N) {
  for (int i=0; i<N; i++) // 1
    hist[buff[i]]++;      // 1
}

int histogram_sum(int *hist, int N) {
  int sum = 0;
  for (int i=0; i<N; i++)
    sum += hist[i];
  return sum;
}


// Each thread computes pairwise product of multiple components of vector.
// Since there are 10 components, but only a maximum of 4 total threads,
// each thread pairwise product of its component, and shifts by a stride
// of the total number of threads. This is done as long as it does not
// exceed the length of the vector. Each thread maintains the sum of the
// pairwise products it calculates.
// 
// Once pairiwise product calculation completes, the per-thread sum is
// stored in a cache, and then all threads in a block sync up to calculate
// the sum for the entire block in a binary tree fashion (in log N steps).
// The overall sum of each block is then stored in an array, which holds
// this partial sum. This partial sum is completed on the CPU. Hence, our
// dot product is complete.
// 
// 1. Compute sum of pairwise product at respective index, while within bounds.
// 2. Shift to the next component, by a stride of total no. of threads (4).
// 3. Store per-thread sum in shared cache (for further reduction).
// 4. Wait for all threads within the block to finish.
// 5. Reduce the sum in the cache to a single value in binary tree fashion.
// 6. Store this per-block sum into a partial sum array.
__global__ void kernel(int *hist, uint8 *buff, int N) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  while (i < N) {
    atomicAdd(&hist[buff[i]], 1);
    i += blockDim.x * gridDim.x;
  }
}


__global__ void kernel_shared(int *hist, uint8 *buff, int N) {
  __shared__ int temp[threads];
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int t = threadIdx.x;
  
  temp[t] = 0;
  __syncthreads();

  while (i < N) {
    atomicAdd(&temp[buff[i]], 1);
    i += blockDim.x * gridDim.x;
  }
  __syncthreads();

  atomicAdd(&hist[t], temp[t]);
}


int run_cpu(uint8* buff, int N) {
  int H = 256;
  int H1 = H * sizeof(int);

  int *hist = (int*) malloc(H1);
  memset(hist, 0, H1);

  clock_t begin = clock();
  histogram(hist, buff, N);
  clock_t end = clock();
  
  double duration = (double) (end - begin) / CLOCKS_PER_SEC;
  printf("CPU execution time: %3.1f ms\n", duration * 1000);
  printf("CPU Histogram sum: %d\n", histogram_sum(hist, H));

  free(hist);
  return 0;
}


int run_gpu(uint8 *buff, int N, int shared) {
  int H = 256;
  int N1 = N * sizeof(uint8);
  int H1 = H * sizeof(int);

  uint8 *buffD;
  int *hist, *histD;
  hist = (int*) malloc(H1);

  hipEvent_t start, stop;
  TRY(hipEventCreate(&start));
  TRY(hipEventCreate(&stop));
  TRY(hipEventRecord(start, 0));

  TRY(hipMalloc(&buffD, N1));
  TRY(hipMemcpy(buffD, buff, N1, hipMemcpyHostToDevice));
  TRY(hipMalloc(&histD, H1));
  TRY(hipMemset(histD, 0, H));

  hipDeviceProp_t p;
  TRY(hipGetDeviceProperties(&p, 0));
  int blocks = 2 * p.multiProcessorCount;
  if (!shared) kernel<<<blocks, threads>>>(histD, buffD, N);
  else kernel_shared<<<blocks, threads>>>(histD, buffD, N);

  float duration;
  TRY( hipMemcpy(hist, histD, H1, hipMemcpyDeviceToHost) );
  TRY( hipEventRecord(stop, 0) );
  TRY( hipEventSynchronize(stop) );
  TRY( hipEventElapsedTime(&duration, start, stop) );
  printf("GPU execution time: %3.1f ms\n", duration);
  printf("GPU Histogram sum: %d\n", histogram_sum(hist, H));

  int* histH = (int*) malloc(H1);
  memset(histH, 0, H1);
  histogram(histH, buff, N);
  int cmp = memcmp(hist, histH, H1);
  if (cmp == 0) printf("GPU Histogram verified.\n");
  else printf("GPU Histogram is wrong!\n");

  TRY(hipEventDestroy(start));
  TRY(hipEventDestroy(stop));
  TRY(hipFree(histD));
  TRY(hipFree(buffD));
  free(histH);
  free(hist);
  return 0;
}


// 1. Allocate space for 2 vectors A, B (of length 10).
// 2. Define vectors A and B.
// 3. Allocate space for partial sum C (of length "blocks").
// 4. Copy A, B from host memory to device memory (GPU).
// 5. Execute kernel with 2 threads per block, and max. 2 blocks (2*2 = 4).
// 6. Wait for kernel to complete, and copy partial sum C from device to host memory.
// 7. Reduce the partial sum C to a single value, the dot product (on CPU).
// 8. Validate if the dot product is correct (on CPU).
int main() {
  int N = 1000000;
  int N1 = N * sizeof(uint8);

  uint8 *buff = (uint8*) malloc(N1);
  memset_rand(buff, N1);

  printf("CPU Histogram ...\n");
  run_cpu(buff, N);
  printf("\n");

  printf("GPU Histogram: atomic ...\n");
  run_gpu(buff, N, 0);
  printf("\n");

  printf("GPU Histogram: shared + atomic ...\n");
  run_gpu(buff, N, 1);
  printf("\n");

  free(buff);
  return 0;
}
